#include "hip/hip_runtime.h"
/*
	ASC - Tema 3
	GPU Hashtable
	Grigore Lucian-Florin
	336CA
*/
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

// load factor boundaries
#define MAX_LOAD_FACTOR .75f
#define MIN_LOAD_FACTOR .55f

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/**
* Hash function based on fnv-1a formulae.
*/
__device__ uint32_t
_hash(int key) {
	uint32_t hash = 2166136261;
    hash = hash ^ ((unsigned char *)(&key))[0];
    hash = hash * 16777619;
    hash = hash ^ ((unsigned char *)(&key))[1];
    hash = hash * 16777619;
    hash = hash ^ ((unsigned char *)(&key))[2];
    hash = hash * 16777619;
    hash = hash ^ ((unsigned char *)(&key))[3];
    hash = hash * 16777619;
    return hash;
}

// computes kernel params
void GpuHashTable::getDeviceParams(int noElems, int &noBlocks, int &noThreads) {
	hipDeviceProp_t deviceProp;

	hipGetDeviceProperties(&deviceProp, 0);
	cudaCheckError();

	noThreads = deviceProp.maxThreadsPerBlock;
	noBlocks = noElems / noThreads;

	if (noBlocks * noThreads != noElems)
		++noBlocks;
}

// returns current load factor
float GpuHashTable::computeLoadFactor() {
	return (float) size / capacity;
}

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size):
	capacity(size), size(0) {
	int noBytes = capacity * sizeof(entry);
	glbGpuAllocator->_cudaMalloc((void **)&hashMap, noBytes);
	cudaCheckError();
	
	hipMemset(hashMap, 0, noBytes);
	cudaCheckError();
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(hashMap);
}

// GPU: copies one entry from old hashmap to a new one based on thread index
__global__ void
kernel_reshape(entry *newHashMap, int newCapacity, entry *oldHashMap, int oldCapacity) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= oldCapacity)
		return;

	int key = oldHashMap[idx].key;
	if (key == 0)
		return;

	int value = oldHashMap[idx].value;
	uint32_t h = _hash(key) % newCapacity;

	// linear probing until a free spot is found
	do {
		if (atomicCAS(&newHashMap[h].key, 0, key) == 0) {
			newHashMap[h].value = value;
			
			return;
		}

		h = (h + 1) % newCapacity;
	} while(1);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	int noBytes = numBucketsReshape * sizeof(entry);

	entry *newHashMap;
	glbGpuAllocator->_cudaMalloc((void **)&newHashMap, noBytes);
	cudaCheckError();

	hipMemset(newHashMap, 0, noBytes);
	cudaCheckError();

	int noBlocks;
	int noThreads;
	getDeviceParams(capacity, noBlocks, noThreads);

	kernel_reshape<<<noBlocks, noThreads>>>(newHashMap, numBucketsReshape, hashMap, capacity);

	hipDeviceSynchronize();
	cudaCheckError();

	glbGpuAllocator->_cudaFree(hashMap);
	cudaCheckError();

	hashMap = newHashMap;
	capacity = numBucketsReshape;
}

// GPU: inserts into hashmap one entry from the given arrays based on thread index
__global__ void
kernel_insert(entry *hashMap, int *keys, int *values, int noKeys, int capacity, int *noUpdates) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= noKeys)
		return;

	int key = keys[idx];
	int value = values[idx];
	uint32_t h = _hash(key) % capacity;
	int prevKey = 0;

	// linear probing until a free spot is found or an update is done
	do {
		prevKey = atomicCAS(&hashMap[h].key, 0, key);

		if (prevKey == 0 || prevKey == key) {
			hashMap[h].value = value;

			if (prevKey == key)
				atomicAdd(noUpdates, 1);

			return;
		}

		h = (h + 1) % capacity;
	} while(1);
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int noBytes = numKeys * sizeof(int);
	int *deviceKeys;
	int *deviceValues;
	int *noUpdates;

	glbGpuAllocator->_cudaMalloc((void **)&deviceKeys, noBytes);
	cudaCheckError();

	glbGpuAllocator->_cudaMalloc((void **)&deviceValues, noBytes);
	cudaCheckError();

	glbGpuAllocator->_cudaMallocManaged((void **)&noUpdates, sizeof(int));
	cudaCheckError();
	*noUpdates = 0;

	hipMemcpy(deviceKeys, keys, noBytes, hipMemcpyHostToDevice);
	cudaCheckError();

	hipMemcpy(deviceValues, values, noBytes, hipMemcpyHostToDevice);
	cudaCheckError();

	if ((size + numKeys) / (float) capacity >= MAX_LOAD_FACTOR)
		reshape((size + numKeys) / MIN_LOAD_FACTOR);

	int noBlocks;
	int noThreads;
	getDeviceParams(numKeys, noBlocks, noThreads);

	kernel_insert<<<noBlocks, noThreads>>>(hashMap, deviceKeys, deviceValues, numKeys, capacity, noUpdates);
	
	hipDeviceSynchronize();
	cudaCheckError();

	size = size + numKeys - *noUpdates;

	glbGpuAllocator->_cudaFree(deviceKeys);
	cudaCheckError();

	glbGpuAllocator->_cudaFree(deviceValues);
	cudaCheckError();

	glbGpuAllocator->_cudaFree(noUpdates);
	cudaCheckError();

	return true;
}

// GPU: returns one value corresponding to one given key based on the thread index
__global__ void
kernel_get(entry *hashMap, int *deviceKeys, int noKeys, int capacity, int *values) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= noKeys)
		return;

	int key = deviceKeys[idx];
	uint32_t h = _hash(key) % capacity;

	// linear probing until the key is found
	do {
		if (key == hashMap[h].key) {
			values[idx] = hashMap[h].value;
			return;
		}

		h = (h + 1) % capacity;
	} while(1);
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *values;
	int *deviceKeys;
	int noBytes = numKeys * sizeof(int);

	glbGpuAllocator->_cudaMalloc((void **)&deviceKeys, noBytes);
	cudaCheckError();

	hipMemcpy(deviceKeys, keys, noBytes, hipMemcpyHostToDevice);
	cudaCheckError();

	glbGpuAllocator->_cudaMallocManaged((void **)&values, noBytes);
	cudaCheckError();

	int noBlocks;
	int noThreads;
	getDeviceParams(numKeys, noBlocks, noThreads);

	kernel_get<<<noBlocks, noThreads>>>(hashMap, deviceKeys, numKeys, capacity, values);

	hipDeviceSynchronize();
	cudaCheckError();

	glbGpuAllocator->_cudaFree(deviceKeys);
	cudaCheckError();

	// return RAM allocated values to not overflow VRAM wrapper - glbGpuAllocator
	int *hostValues = (int *) malloc(noBytes);
	hipMemcpy(hostValues, values, noBytes, hipMemcpyDeviceToHost);
	cudaCheckError();

	glbGpuAllocator->_cudaFree(values);
	cudaCheckError();

	return hostValues;
}
