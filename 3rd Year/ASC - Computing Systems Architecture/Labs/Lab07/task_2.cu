#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
		c[i] = a[i] + b[i];
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;
	hipError_t err;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays
	
	host_array_a = (float *) malloc(N * sizeof(float));
	host_array_b = (float *) malloc(N * sizeof(float));
	host_array_c = (float *) malloc(N * sizeof(float));

    // TODO 2: Allocate the device's arrays
	
	err = hipMalloc(&device_array_a, N * sizeof(float));
	DIE(err != hipSuccess, "hipMalloc err");
	err = hipMalloc(&device_array_b, N * sizeof(float));
	DIE(err != hipSuccess, "hipMalloc err");
	err = hipMalloc(&device_array_c, N * sizeof(float));
	DIE(err != hipSuccess, "hipMalloc err");

    // TODO 3: Check for allocation errors
	
	DIE(host_array_a == NULL || host_array_b == NULL || host_array_c == NULL, "malloc err");

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.

	fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);

    // TODO 5: Copy the host's arrays to device
	
	err = hipMemcpy(device_array_a, host_array_a, N * sizeof(float), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "first cudamemcpy");

	err = hipMemcpy(device_array_b, host_array_b, N * sizeof(float), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "second cudamemcpy");

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).
	const size_t block_size = 256;
	size_t no_blocks = N / block_size;

	if (N % block_size != 0)
		++no_blocks;

	add_arrays<<<no_blocks, block_size>>>(device_array_a, device_array_b, device_array_c, N);

    // TODO 7: Copy back the results and then uncomment the checking function

	err = hipDeviceSynchronize();
	DIE(err != hipSuccess, "cudadevicesynchronize err");

	err = hipMemcpy(host_array_c, device_array_c, N * sizeof(float), hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "last hipMemcpy err");

    check_task_2(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory
 
	free(host_array_a);
	free(host_array_b);
	free(host_array_c);
	
	err = hipFree(device_array_a);
	DIE(err != hipSuccess, "hipFree err");
	err = hipFree(device_array_b);
	DIE(err != hipSuccess, "hipFree err");
	err = hipFree(device_array_c);
	DIE(err != hipSuccess, "hipFree err");

    return 0;
}
