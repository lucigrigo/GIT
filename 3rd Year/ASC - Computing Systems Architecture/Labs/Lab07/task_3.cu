#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

int main(void) {
    hipSetDevice(0);

    int *host_array_a = 0;
    int *host_array_b = 0;

    int *device_array_a = 0;
    int *device_array_b = 0;
    int *device_array_c = 0;

    // TODO 1: Allocate the host's arrays with the specified number of elements:
    // host_array_a => 32M
    // host_array_b => 32M
	host_array_a = (int *) malloc(BUF_32M * sizeof(int));
	host_array_b = (int *) malloc(BUF_32M * sizeof(int));

    // TODO 2: Allocate the device's arrays with the specified number of elements:
    // device_array_a => 32M
    // device_array_b => 32M
    // device_array_c => 2M
	hipMalloc(&device_array_a, BUF_32M * sizeof(float));
	hipMalloc(&device_array_b, BUF_32M * sizeof(float));
	hipMalloc(&device_array_c, BUF_2M * sizeof(float));

    // Check for allocation errors
    if (host_array_a == 0 || host_array_b == 0 || 
        device_array_a == 0 || device_array_b == 0 || 
        device_array_c == 0) {
        printf("[*] Error!\n");
        return 1;
    }

    for (int i = 0; i < BUF_32M; ++i) {
        host_array_a[i] = i % 32;
        host_array_b[i] = i % 2;
    }

    printf("Before swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 3: Copy from host to device

	hipError_t err = hipMemcpy(device_array_a, host_array_a, BUF_32M * sizeof(int), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "1st cudamemcpy");

	err = hipMemcpy(device_array_b, host_array_b, BUF_32M * sizeof(int), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "2nd cudamemcpy");

    // TODO 4: Swap the buffers (BUF_2M values each iteration)
    // Hint 1: device_array_c should be used as a temporary buffer
    // Hint 2: hipMemcpy

	for (int i = 0; i != BUF_32M; i += BUF_2M) {
		err = hipMemcpy(device_array_c, device_array_b + i,
						BUF_2M * sizeof(int),
						hipMemcpyDeviceToDevice);
		DIE(err != hipSuccess, "1st for cudamemcpy");

		err = hipMemcpy(device_array_b + i, device_array_a + i,
						BUF_2M * sizeof(int),
						hipMemcpyDeviceToDevice);
		DIE(err != hipSuccess, "2nd for cudamemcpy");

		err = hipMemcpy(device_array_a + i, device_array_c,
						BUF_2M * sizeof(int),
						hipMemcpyDeviceToDevice);
		DIE(err != hipSuccess, "3rd for cudamemcpy");
	}

    // TODO 5: Copy from device to host

	err = hipMemcpy(host_array_a, device_array_a,
					BUF_2M * sizeof(int), hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "first cudamemcpy after");

	err = hipMemcpy(host_array_b, device_array_b,
					BUF_2M * sizeof(int), hipMemcpyDeviceToHost);
	DIE(err != hipSuccess, "second cudamemcpy after");

    printf("\nAfter swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 6: Free the memory

	free(host_array_a);
	free(host_array_b);

	err = hipFree(device_array_a);
	DIE(err != hipSuccess, "first cudafree");

	err = hipFree(device_array_b);
	DIE(err != hipSuccess, "second cudafree");

	err = hipFree(device_array_c);
	DIE(err != hipSuccess, "third cudafree");

    return 0;
}
